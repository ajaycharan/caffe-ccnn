#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layer_factory.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void HardmaxLayer_Forward_dev(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype maxval = data[(n * channels + 0) * spatial_dim + s];
    int maxind = 0;
    for (int c = 0; c < channels; ++c) {
      Dtype v = data[(n * channels + c) * spatial_dim + s];
      if (v > maxval) {
        maxval = v;
        maxind = c;
      }
    }
    out[(n * channels + maxind) * spatial_dim + s] = 1;
  }
}

template<typename Dtype>
void HardmaxLayer_Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int num = top[0]->num();
  int channels = top[0]->channels();
  int spatial_dim = top[0]->height()*top[0]->width();
  caffe_gpu_set(top[0]->count(), Dtype(0), top_data);
  // NOLINT_NEXT_LINE(whitespace/operators)
  HardmaxLayer_Forward_dev<Dtype><<<CAFFE_GET_BLOCKS(num * spatial_dim),
      CAFFE_CUDA_NUM_THREADS>>>(num, channels, spatial_dim, bottom_data,
      top_data);
}
template void HardmaxLayer_Forward_gpu<float>(const vector<Blob<float>*>&,
  const vector<Blob<float>*>& top);
template void HardmaxLayer_Forward_gpu<double>(const vector<Blob<double>*>&,
  const vector<Blob<double>*>& top);
} // Namespace caffe